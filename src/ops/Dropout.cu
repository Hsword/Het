#include "hip/hip_runtime.h"
#include "gpu_runtime.h"
#include <hiprand.h>
// #include <hiprand/hiprand_kernel.h>

__global__ void dropout_kernel(const float *input, float *output, const float rate, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size) return;
    float keep_mask = (float)(output[ind] >= rate);
    output[ind] = input[ind] * keep_mask / (1 - rate);
}

int DLGpuDropout(const DLArrayHandle input, const float dropout, DLArrayHandle output, unsigned long long* pseed, DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    const float *input_data = (const float *)input->data;
    float *output_data = (float *)output->data;

    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
    *pseed = time(0);
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, *pseed));
    CURAND_CALL(hiprandGenerateUniform(gen, output_data, size));
    CURAND_CALL(hiprandDestroyGenerator(gen));

    dim3 blocks;
    dim3 threads;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle) {
        dropout_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(input_data, output_data, dropout, size);
    } else {
        dropout_kernel<<<blocks, threads>>>(input_data, output_data, dropout, size);
    }
    return 0;
}

int DLGpuDropoutGradient(const DLArrayHandle grad, const float dropout, DLArrayHandle output, unsigned long long seed, DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < grad->ndim; i++) {
        size *= grad->shape[i];
    }
    const float *grad_data = (const float *)grad->data;
    float *output_data = (float *)output->data;

    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
    CURAND_CALL(hiprandGenerateUniform(gen, output_data, size));
    CURAND_CALL(hiprandDestroyGenerator(gen));

    dim3 blocks;
    dim3 threads;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle) {
        dropout_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(grad_data, output_data, dropout, size);
    } else {
        dropout_kernel<<<blocks, threads>>>(grad_data, output_data, dropout, size);
    }
    return 0;
}
