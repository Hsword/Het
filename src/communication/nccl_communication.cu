#include "hip/hip_runtime.h"
#include "../header/nccl_communication.h"
__global__ void array_set(float *a, float tmp, int size){
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id >= size) return;
  a[id] = tmp;
}

void create_streams(hipStream_t *stream, int *devices, int devices_numbers){
  // stream = (hipStream_t *) stream;
  // *stream = (hipStream_t*)malloc(sizeof(hipStream_t) * devices_numbers);
  for(int i = 0; i < devices_numbers; i++){
		CUDACHECK(hipSetDevice(devices[i]));
		CUDACHECK(hipStreamCreate((hipStream_t *)stream + i));
  }
}

/*
void update_stream(size_t dev_id, hipStream_t *stream, DLStreamHandle stream_handle){
  stream[dev_id] = *(hipStream_t*)stream_handle->handle;
}
*/
void update_stream(size_t dev_id, hipStream_t *stream, hipStream_t *stream_handle){
  stream[dev_id] = *stream_handle;
}

void free_streams(hipStream_t *stream, int *devices, int devices_numbers){
	for(int i = 0; i < devices_numbers; i++){
		CUDACHECK(hipSetDevice(devices[i]));
		CUDACHECK(hipStreamDestroy(stream[i]));
	}
}

void init_NCCL(ncclComm_t *comms, int *devices, int devices_numbers){
  // *comms = (ncclComm_t *)malloc(sizeof(ncclComm_t) * devices_numbers);
  NCCLCHECK(ncclCommInitAll(comms, devices_numbers, devices));
}

void finish_NCCL(ncclComm_t *comms, int devices_numbers){
  for(int i = 0; i < devices_numbers; i++)
    NCCLCHECK(ncclCommDestroy(comms[i]));
}

void Synchronize_streams(hipStream_t *stream, int *devices, int devices_numbers){
  for(int i = 0; i < devices_numbers; i++){
    CUDACHECK(hipSetDevice(devices[i]));
    CUDACHECK(hipStreamSynchronize(stream[i]));
  }
}

void NCCL_AllReduce(float** sendbuff, float** recvbuff, int size, 
        ncclComm_t *comms, hipStream_t *stream, int devices_numbers){
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < devices_numbers; ++i)
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
        comms[i], stream[i]));
  NCCLCHECK(ncclGroupEnd());
}
void display(const float *device_data, int dev_id, int size){
  printf("Display Device %d:\n", dev_id);
  CUDACHECK(hipSetDevice(dev_id));
  float *host_buff;
  CUDACHECK(hipHostAlloc(&host_buff, size * sizeof(float), hipHostMallocDefault));
  CUDACHECK(hipMemcpy(host_buff, device_data, size * sizeof(float), hipMemcpyDeviceToHost));
  for(int i = 0; i < size; i++){
    printf("%f ",host_buff[i]);
  }
  printf("\n");
  CUDACHECK(hipHostFree(host_buff));
}

void create(int **a, int n){
  *a = (int *)malloc(sizeof(int) * n);
  for(int i = 0; i < n; i++){
    (*a)[i] = 1;
  }
}
void for_each(int *a, int n){
  for(int i = 0; i < n; i++){
    printf("%d ",a[i]);
  }
  printf("\n");
}
void show_int(int a){
  printf("the num is %d\n", a);
}

void show_array2D(float **a, int row, int col){
  for(int i = 0; i < row; i++){
    for(int j = 0; j < col; j++){
      printf("%f ", a[i][j]);
    }
    printf("\n");
  }
}

int main(){

  // int nDev = 8;
  // int size = 16;
  // int devs[8] = {0, 1, 2, 3, 4, 5, 6, 7};
  // hipStream_t *s;
  // ncclComm_t *comms;

  // //allocating and initializing device buffers
  // float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  // float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  // // hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  // for(int i = 0; i < nDev; ++i) {
  //   CUDACHECK(hipSetDevice(devs[i]));
  //   CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
  //   CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
  // }

  // create_streams(&s, devs, nDev);
  
  // //init_data
  // for(int i = 0; i < nDev; i++){
	// 	CUDACHECK(hipSetDevice(i));
	// 	array_set<<<(size + THREADS_PER_BLOCKS - 1)/THREADS_PER_BLOCKS, THREADS_PER_BLOCKS, 0, s[i]>>>(sendbuff[i], i, size);
  //   array_set<<<(size + THREADS_PER_BLOCKS - 1)/THREADS_PER_BLOCKS, THREADS_PER_BLOCKS, 0, s[i]>>>(recvbuff[i], 0, size);
  // }

  // init_NCCL(&comms, devs, nDev);
  // NCCL_AllReduce(sendbuff, recvbuff, size, comms, s, nDev);
  // Synchronize_streams(s, devs, nDev);
  
  // for(int i = 0; i < nDev; i++){
  //   display(recvbuff[i], devs[i], size);
  // }
  printf("comm_t size is %ld\n", sizeof(ncclComm_t*));
  printf("void* size is %ld\n", sizeof(void*));
  return 0;
}

